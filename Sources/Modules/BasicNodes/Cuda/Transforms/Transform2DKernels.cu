#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C"  
{
	__global__ void BilinearResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;
		int size =  outputWidth * outputHeight;

		if (id < size) 
		{
			int px = id % outputWidth;
			int py = id / outputWidth;

			float xRatio = (float)(inputWidth - 1) / (outputWidth);
			float yRatio = (float)(inputHeight - 1) / (outputHeight);

			int x = (int) (xRatio * (px+.5f));
			int y = (int) (yRatio * (py+.5f));          
 
			// X and Y distance difference
			float xDist = (xRatio * (px+.5f)) - x+.5f;
			float yDist = (yRatio * (py+.5f)) - y+.5f;
 
			// Points
			float topLeft = input[y * inputWidth + x];
			float topRight = input[y * inputWidth + x + 1];
			float bottomLeft = input[(y + 1) * inputWidth + x];
			float bottomRight = input[(y + 1) * inputWidth + x + 1]; 
                
			float result = 
				topLeft * (1 - xDist) * (1 - yDist) + 
				topRight * xDist * (1 - yDist) + 
				bottomLeft * yDist * (1 - xDist) + 
				bottomRight * xDist * yDist;
 
			output[py * outputWidth + px] = result;
		}
	}




	__global__ void NNResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;
		int size =  outputWidth * outputHeight;

		if (id < size) 
		{
			int px = id % outputWidth;
			int py = id / outputWidth;

			float xRatio = (float)(inputWidth - 1) / (outputWidth);
			float yRatio = (float)(inputHeight - 1) / (outputHeight);

			int x = (int) (xRatio * (px+.5f));
			int y = (int) (yRatio * (py+.5f));           

			output[py * outputWidth + px] = input[y*inputWidth + x];
		}
	}




	__global__ void BilinearResampleSubImageKernel(float *input, float *output, float* subImageDefs, bool safeBounds,
		int inputWidth, int inputHeight, int outputWidth, int outputHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;
		int size =  outputWidth * outputHeight;

		if (id < size) 
		{
			float subImgCX = subImageDefs[0]; // <-1, 1>
			float subImgCY = subImageDefs[1]; // <-1, 1>
			float subImgDiameter = subImageDefs[2]; // <0,1>

			int maxDiameter = min(inputWidth - 1, inputHeight - 1);
			int diameterPix = (int)(subImgDiameter * maxDiameter);

			diameterPix = max(1, diameterPix);
			diameterPix = min(maxDiameter, diameterPix);

			int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
			int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

			if (safeBounds) 
			{
				subImgX = max(subImgX, 1);
				subImgY = max(subImgY, 1);

				subImgX = min(subImgX, inputWidth - diameterPix - 1);
				subImgY = min(subImgY, inputHeight - diameterPix - 1);			
			}

			int px = id % outputWidth;
			int py = id / outputWidth;
				
			float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
			float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

			int x = (int) (xRatio * px);
			int y = (int) (yRatio * py);   

			if (x + subImgX >= 0 && y + subImgY >= 0 &&
				x + subImgX < inputWidth && y + subImgY < inputHeight) 
			{
				// X and Y distance difference
				float xDist = (xRatio * px) - x;
				float yDist = (yRatio * py) - y;
 
				// Points
				float topLeft= input[(y + subImgY) * inputWidth + x + subImgX];
				float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
				float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
				float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1]; 
                
				float result = 
					topLeft * (1 - xDist) * (1 - yDist) + 
					topRight * xDist * (1 - yDist) + 
					bottomLeft * yDist * (1 - xDist) + 
					bottomRight * xDist * yDist;
 
				output[py * outputWidth + px] = result;
			}
		}
	}




    ///  Resmaple for the set of locations. It needs proper 
   	__global__ void BilinearResampleSubImageKernel_ForManyProposals(const float *input, float *output, const float* subImageDefs, bool safeBounds,
		int subImageDefsDim, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int numberSubImages, int outputSize)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;

        int px = id % outputWidth;  // line in the single output image
        int subim_id = id / outputWidth / outputHeight;  // which image it is
        int py = (id / outputWidth) % outputHeight;  // column in the single output image

        if (id<outputSize)
        {
			float subImgCX = subImageDefs[0 + subim_id*subImageDefsDim]; // <-1, 1>
			float subImgCY = subImageDefs[1 + subim_id*subImageDefsDim]; // <-1, 1>
			float subImgDiameter = subImageDefs[2 + subim_id*subImageDefsDim]; // <0,1>

			int maxDiameter = min(inputWidth - 1, inputHeight - 1);
			int diameterPix = (int)(subImgDiameter * maxDiameter);

			diameterPix = max(1, diameterPix);
			diameterPix = min(maxDiameter, diameterPix);

			int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
			int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

			if (safeBounds) 
			{
				subImgX = max(subImgX, 1);
				subImgY = max(subImgY, 1);

				subImgX = min(subImgX, inputWidth - diameterPix - 1);
				subImgY = min(subImgY, inputHeight - diameterPix - 1);			
			}

			float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
			float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

			int x = (int) (xRatio * px);
			int y = (int) (yRatio * py);   

			if (x + subImgX >= 0 && y + subImgY >= 0 &&
				x + subImgX < inputWidth && y + subImgY < inputHeight) 
			{
				//--- X and Y distance difference
				float xDist = (xRatio * px) - x;
				float yDist = (yRatio * py) - y;
 
				//--- Points
				float topLeft= input[(y + subImgY) * inputWidth + x + subImgX];
				float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
				float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
				float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1 ]; 
                
				float result = 
					topLeft * (1 - xDist) * (1 - yDist) + 
					topRight * xDist * (1 - yDist) + 
					bottomLeft * yDist * (1 - xDist) + 
					bottomRight * xDist * yDist;
 
				output[py * outputWidth + px + subim_id*outputWidth*outputHeight] = result;
			}
        }
	}





	__global__ void BilinearAddSubImageKernel(float *input, float *opImage, float* subImageDefs, int inputWidth, int inputHeight, int opImageWidth, int opImageHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;		

		float subImgCX = subImageDefs[0]; // <-1, 1>
		float subImgCY = subImageDefs[1]; // <-1, 1>
		float subImgDiameter = subImageDefs[2]; // <0,1>

		int maxDiameter = min(inputWidth, inputHeight);
		int diameterPix = (int)(subImgDiameter * maxDiameter);
		diameterPix = max(1, diameterPix);

		int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
		int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

		int px = id % diameterPix;
		int py = id / diameterPix;

		if (px + subImgX >= 0 && py + subImgY >= 0 &&
			px + subImgX < inputWidth && py + subImgY < inputHeight &&
			py < diameterPix ) 
		{						
			float xRatio = (float)(opImageWidth - 1) / (diameterPix);
			float yRatio = (float)(opImageHeight - 1) / (diameterPix);

			int x = (int) (xRatio * px);
			int y = (int) (yRatio * py);          
 
			// X and Y distance difference
			float xDist = (xRatio * px) - x;
			float yDist = (yRatio * py) - y;
 
			// Points
			float topLeft= opImage[y * opImageWidth + x];
			float topRight = opImage[y * opImageWidth + x + 1];
			float bottomLeft = opImage[(y + 1) * opImageWidth + x];
			float bottomRight = opImage[(y + 1) * opImageWidth + x + 1]; 
                
			float result = 
				topLeft * (1 - xDist) * (1 - yDist) + 
				topRight * xDist * (1 - yDist) + 
				bottomLeft * yDist * (1 - xDist) + 
				bottomRight * xDist * yDist;
				
  
			input[(py + subImgY) * inputWidth + px + subImgX] += result;
		}
	}

	__global__ void DrawSpriteKernel(float *input, int inputWidth, int inputHeight, float *sprite, float2 position, int2 spriteSize)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;

		int inputSize = inputWidth * inputHeight;
		int size = spriteSize.x * spriteSize.y;
		
		int px = id % spriteSize.x;
		int py = id / spriteSize.x;

		int inputOffset = ((int)position.y + py) * inputWidth + position.x + px;

		if (id < size && inputOffset >= 0 && inputOffset < inputSize) 
		{
			input[inputOffset] = sprite[id];
		}
	}

	__global__ void Crop2DKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int size, int leftMargin, int topMargin, float fillValue)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
				+ blockDim.x * blockIdx.x
				+ threadIdx.x;

		if (id < size) 
		{
			int inputX = id % outputWidth - leftMargin;
			int inputY = id / outputWidth - topMargin;

			if (inputX >= 0 && inputX < inputWidth && inputY >= 0 && inputY < inputHeight)
				output[id] = input[inputX + inputY * inputWidth];
			else
				output[id] = fillValue;
		}
	}





    //------------------------------------------------------------------------------------------------------------------------
    //                          RETINA STUFF
    //------------------------------------------------------------------------------------------------------------------------

    __device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds,
		int inputWidth, int inputHeight,
        int2 & subImg, int & diameterPix)
    {
    	diameterPix = (int)( fminf( (float)inputWidth,(float)inputHeight ) * subImageDefs[2] ); // <0,1> 

		subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f) ;//- diameterPix / 2;
		subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);// - diameterPix / 2;

		int maxDiameter = min(inputWidth - 1, inputHeight - 1);

        diameterPix = max(1, diameterPix);
		diameterPix = min(maxDiameter, diameterPix);

		if (safeBounds) 
		{
			subImg.x = max(subImg.x, 1);
			subImg.y = max(subImg.y, 1);
			subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
			subImg.y = min(subImg.y, inputHeight - diameterPix - 1);			
		}
    }


    __global__ void RetinaTransform_HaveAtLeastOneValueThere (float * subImageDefs, 
                                                     float* input, int inputWidth, int inputHeight,
                                                     float* output,int outputDataSize,
                                                     float* retinaMask, int retinaDataSize, int retinaMaskColHint,
                                                     float* retinaDataInserted)
    {
        int id_retinaPoint = blockDim.x * blockIdx.y * gridDim.x
				    + blockDim.x * blockIdx.x
				    + threadIdx.x;

		int2 subImg;
        int diameterPix;
        bool  safeBounds = 0;


        EstimateParForSubsample( subImageDefs,  safeBounds, inputWidth,  inputHeight,  subImg, diameterPix );

        if (id_retinaPoint<outputDataSize)
        {
            output[id_retinaPoint] = 0; // default value
            float x_mask = (retinaMask[id_retinaPoint*retinaMaskColHint]*diameterPix);
            float y_mask = (retinaMask[id_retinaPoint*retinaMaskColHint+1]*diameterPix);

            int x = subImg.x + x_mask;
            int y = subImg.y + y_mask;
            if (x<inputWidth && y<inputHeight && x>=0 && y>=0)
            {
                float val = input[x+y*inputWidth];
                output[id_retinaPoint] = val;

                atomicAdd(output + id_retinaPoint , val);
                atomicAdd(retinaDataInserted + id_retinaPoint , 1);
            }
        }
    }

    __global__ void RetinaTransform_FillRetinaAtomic (float * subImageDefs, 
                                                       float* input, int inputWidth, int inputHeight,
                                                       float* output,int outputDataSize,
                                                       float* retinaMask, int retinaDataSize, int retinaMaskColHint,
                                                       float* retinaDataInserted)
    {
        int id_pxl = blockDim.x * blockIdx.y * gridDim.x
				    + blockDim.x * blockIdx.x
				    + threadIdx.x;

		int2 subImg;
        int diameterPix;
        bool  safeBounds = 0;

        int x = id_pxl % inputWidth;
        int y = id_pxl/inputWidth;

        EstimateParForSubsample( subImageDefs,  safeBounds, inputWidth,  inputHeight,  subImg, diameterPix );

        if (id_pxl<inputWidth*inputHeight)
        {
            float minDist = 999999.9; // ??>? should be written bette
            int minIdx = 1;
            for (int id_retinaPoint=0 ; id_retinaPoint<retinaDataSize ; id_retinaPoint++)
            {
                float x_mask = (retinaMask[id_retinaPoint*retinaMaskColHint]*diameterPix);
                float y_mask = (retinaMask[id_retinaPoint*retinaMaskColHint+1]*diameterPix);

                x_mask += subImg.x;
                y_mask += subImg.y;

                float dist = (x-x_mask)*(x-x_mask) + (y-y_mask)*(y-y_mask);

                if (dist<minDist)
                {
                    minDist = dist;
                    minIdx  = id_retinaPoint;
                }
            }
            atomicAdd(output + minIdx , input[id_pxl]);
            atomicAdd(retinaDataInserted + minIdx , 1);
        }
    }



    

}

