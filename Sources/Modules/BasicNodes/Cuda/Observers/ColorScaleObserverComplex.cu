#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <hip/hip_runtime_api.h>
#include <> 
#include <> 
#include <builtin_types.h> 
#include <hip/hip_vector_types.h> 
#include <vector_functions.h> 
#include <float.h>

#include "ColorHelpers.cu"

extern "C"
{
	__global__ void ColorScaleObserverComplex(float* values, int method, int scale, float minValue, float maxValue, unsigned int* pixels, int numOfPixels)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x
			+ blockDim.x*blockIdx.x
			+ threadIdx.x;

		if (id < numOfPixels) //id of the thread is valid
		{
			pixels[id] = float_to_uint_rgba(values[id], method, scale, minValue, maxValue);
		}
	}
}