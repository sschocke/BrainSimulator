#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include "ColorScaleObserverSingle.cu"
#include "../Common/Statistics.cu"
#include "../Transforms/Transform2DKernels.cu"




extern "C"  
{

	//kernel code
    // There is ,,Vision/KMeansWM.cu/FocuserInputObserver''   that defines part of this function as device and has some features as plotting multiple lines 
	__global__ void FocuserInputObserver(float* values, float* pupilControl, int inputWidth, int inputHeight, unsigned int* pixels)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		int numOfPixels = inputWidth * inputHeight;

		if(id < numOfPixels) //id of the thread is valid
		{		
			float cX = pupilControl[0]; // <-1, 1>
			float cY = pupilControl[1]; // <-1, 1>
			float subImgDiameter = pupilControl[2]; // <0,1>

			int maxDiameter = min(inputWidth, inputHeight);
			int diameterPix = (int)(subImgDiameter * maxDiameter);
			diameterPix = max(1, diameterPix);

			int cXPix = (int)(inputWidth * (cX + 1) * 0.5f);
			int cYPix = (int)(inputHeight * (cY + 1) * 0.5f);

			int subImgX = cXPix - diameterPix / 2;
			int subImgY = cYPix - diameterPix / 2;

			subImgX = max(subImgX, 0);
			subImgY = max(subImgY, 0);

			subImgX = min(subImgX, inputWidth - diameterPix);
			subImgY = min(subImgY, inputHeight - diameterPix);

			int px = id % inputWidth;
			int py = id / inputWidth;			

			float hue = 1.0f;
			float saturation = 0;				
			float value = values[id];			

			if (px >= subImgX && py >= subImgY && 
				px <= subImgX + diameterPix && py <= subImgY + diameterPix) 
			{
				saturation = 0.33f;
				value += 0.2f;
			}			

			if (px == cXPix || py == cYPix) 
			{
				saturation = 1.0f;
				value = 0.7f;
			}			

			value = fminf(fmaxf(value, 0), 1);

			pixels[id] = hsva_to_uint_rgba(hue, saturation, value, 1.0f);
		}
	}

	__constant__ int NUM_C_VALUES = 5;

	__global__ void PupilControlObserver(float* values, Centroid* centroids, int centroidsCount, int inputWidth, int inputHeight, unsigned int* pixels)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		int numOfPixels = inputWidth * inputHeight;

		if(id < numOfPixels) //id of the thread is valid
		{			
			int px = id % inputWidth;
			int py = id / inputWidth;			

			float hue = 0.6f;
			float saturation = 0;				
			float value = values[id];			
			
			for (int i = 0; i < centroidsCount; i++) 
			{

				float cX = centroids[i].X; // <-1, 1>
				float cY = centroids[i].Y; // <-1, 1>		

				float cStdDevX = centroids[i].VarianceX;
				float cStdDevY = centroids[i].VarianceY;

				int cXPix = (int)(inputWidth * (cX + 1) * 0.5f);
				int cYPix = (int)(inputHeight * (cY + 1) * 0.5f);

				int cStdDevXPix = (int)(inputWidth * cStdDevX);
				int sStdDevYPix = (int)(inputHeight * cStdDevY);

				if (px >= cXPix - cStdDevXPix && py >= cYPix - sStdDevYPix && 
					px <= cXPix + cStdDevXPix && py <= cYPix + sStdDevYPix) 
				{
					hue = 0.33;
					saturation = 0.5;
					value += 0.2;
				}

				if (px >= cXPix - 2 && py >= cYPix - 2 && 
					px <= cXPix + 2 && py <= cYPix + 2) 
				{
					hue = 0.6;
					saturation = 1;
					value = 1;
				}					
			}
			
			value = fminf(fmaxf(value, 0), 1);

			pixels[id] = hsva_to_uint_rgba(hue, saturation, value, 1.0f);
		}
	}



    //------------------------------------------------------------------------------------------------------------------------
    //                          RETINA STUFF
    //------------------------------------------------------------------------------------------------------------------------

    __global__ void RetinaObserver_Mask(unsigned int* pixels, int pixelsWidth, int pixelsHeight, float* retinaPtsDefs, int retinaPtsDefsSize, float* subImageDefs)
    {
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;
        float x,y;        

        int2 subImg;
        int diameterPix;
        bool  safeBounds = 0;

        EstimateParForSubsample( subImageDefs,  safeBounds,		pixelsWidth,  pixelsHeight,       subImg, diameterPix);

        if (id < retinaPtsDefsSize)
        {
            x = (float)subImg.x + (retinaPtsDefs[id * 2]*diameterPix);
            y = (float)subImg.y + (retinaPtsDefs[id * 2 + 1]*diameterPix);

            if (x>0 && y>0 && x<pixelsWidth && y<pixelsHeight)
                pixels[(int)x+(int)y*pixelsWidth] = GET_RGBA(0,255,0,255);
        }
    }


    // next code should be improved:
    //         - share memory for minDist
    //         - template specialization to have it just once.  It does not work now :(

    __global__ void RetinaObserver_UnMaskPatchFl(float* output2save, int pixelsWidth, int pixelsHeight, float* retinaPtsDefs, int retinaPtsDefsSize, float* retinaValues, float* subImageDefs)
    {
      int id_pxl = blockDim.x * blockIdx.y * gridDim.x
		          + blockDim.x * blockIdx.x
        	      + threadIdx.x;

		int2 subImg;
        int diameterPix;
        bool  safeBounds = 0;

        int x = id_pxl % pixelsWidth;
        int y = id_pxl / pixelsWidth;

        EstimateParForSubsample( subImageDefs,  safeBounds, pixelsWidth,  pixelsHeight,  subImg, diameterPix );

        if (id_pxl < pixelsWidth*pixelsHeight)
        {
            float minDist = 999999.9f;
            int minIdx = 0;
            for (int i = 0; i < retinaPtsDefsSize; i++)
            {
                float xr = subImg.x + retinaPtsDefs[i * 2]*(float)diameterPix;
                float yr = subImg.y + retinaPtsDefs[i * 2 + 1]*(float)diameterPix;
                float dist = (xr - x) * (xr - x) + (yr - y) * (yr - y);
                if (dist < minDist)
                {
                    minDist = dist;
                    output2save[id_pxl] = fminf(fmaxf(retinaValues[i],0.0f),1.0f);
                }
            }
        }
    }

    __global__ void RetinaObserver_UnMaskPatchVBO(unsigned int* pixels, int pixelsWidth, int pixelsHeight, float* retinaPtsDefs, int retinaPtsDefsSize,  float* retinaValues, float* subImageDefs)
    {
      int id_pxl = blockDim.x * blockIdx.y * gridDim.x
		          + blockDim.x * blockIdx.x
        	      + threadIdx.x;

		int2 subImg;
        int diameterPix;
        bool  safeBounds = 0;

        int x = id_pxl % pixelsWidth;
        int y = id_pxl / pixelsWidth;

        EstimateParForSubsample( subImageDefs,  safeBounds, pixelsWidth,  pixelsHeight,  subImg, diameterPix );

        if (id_pxl < pixelsWidth*pixelsHeight)
        {
            float minDist = 999999.9f;
            int minIdx = 0;
            for (int i = 0; i < retinaPtsDefsSize; i++)
            {
                float xr = subImg.x + retinaPtsDefs[i * 2]*(float)diameterPix;
                float yr = subImg.y + retinaPtsDefs[i * 2 + 1]*(float)diameterPix;
                float dist = (xr - x) * (xr - x) + (yr - y) * (yr - y);
                if (dist < minDist)
                {
                    minDist = dist;
                    pixels[id_pxl] = hsva_to_uint_rgba(0.5f, 1.0f, fminf(fmaxf(retinaValues[i],0.0f),1.0f), 1.0f);
                }
            }
        }
    }



}

