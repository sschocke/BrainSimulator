#include <hip/hip_runtime.h>


/*
Inspired by the implementation of CustomPong.cu + GridWorld.cu

@author mp
*/
extern "C"
{

	/*
	Draws a background color into a 3-component image.
	inputWidth & inputHeight: map dimensions in pixels
	gridDim.y = 3, one for each color component
	*/
	__global__ void DrawRgbBackgroundKernel(float *target, int inputWidth, int inputHeight,
		float r, float g, float b)
	{
		int column = threadIdx.x + blockDim.x * blockIdx.z;
		if (column >= inputWidth)
			return;

		int id = inputWidth * ( blockIdx.y * gridDim.x + blockIdx.x) // blockIdx.x == row, blockIdx.y == color channel 
			+ column;

		int imagePixels = inputWidth * inputHeight; 

		if (id < 3*imagePixels) // 3 for RGB 
		{
			float color = 0.0f;
			switch (blockIdx.y)
			{
			case 0:
				color = r;
				break;
			case 1:
				color = g;
				break;
			case 2:
				color = b;
				break;
			}
			target[id] = color;
		}
	}

	/*
	Adds noise into a 3-component image.
	inputWidth & inputHeight: map dimensions in pixels
	*/
	__global__ void AddRgbNoiseKernel(float *target, int inputWidth, int inputHeight, float *randoms)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int imagePixels = inputWidth * inputHeight;

		if (id < imagePixels)
		{
			unsigned int tg = *((unsigned int*)(&target[id]));

			int blue = (tg >> 0) & (0xFF);
			blue += (int)(randoms[id]);
			blue = blue < 255 ? blue : 255;
			blue = blue > 0 ? blue : 0;

			int green = ((tg >> 8) & (0xFF));
			green += (int)(randoms[id + imagePixels]);
			green = green < 255 ? green : 255;
			green = green > 0 ? green : 0;

			int red = ((tg >> 16) & (0xFF));
			red += (int)(randoms[id + imagePixels * 2]);
			red = red < 255 ? red : 255;
			red = red > 0 ? red : 0;

			// alpha is the last channel (<< 24)
			unsigned int tmp = (*((unsigned int *)(&blue)) << 0)
						     | (*((unsigned int *)(&green)) << 8)
							 | (*((unsigned int *)(&red)) << 16);

			target[id] = *((float *)(&tmp));
		}
	}

	/* Fill specified rectangle with color */
	__global__ void DrawRgbaColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		int areaWidth, int areaHeight, float r, float g, float b)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = areaWidth * areaHeight;

		int idTextureRgb = id / texturePixels;
		int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
		int idTextureY = idTexturePixel / areaWidth;
		int idTextureX = (idTexturePixel - idTextureY * areaWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


		if (idTextureRgb < 3) // 3 channels that we will write to
		{
			// if the texture pixel offset by inputX, inputY, lies inside the target
			if (idTextureX + inputX < targetWidth &&
				idTextureX + inputX >= 0 &&
				idTextureY + inputY < targetHeight &&
				idTextureY + inputY >= 0)
			{
				float color = 0.0f;
				switch (idTextureRgb)
				{
				case 0:
					color = r;
					break;
				case 1:
					color = g;
					break;
				case 2:
					color = b;
					break;
				}
				int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
				target[tIndex] = color;
			}
		}
	}

	/*
	Draws a texture into a 3-component target. RGBA. Checks bounds.
	*/
	__global__ void DrawRgbaTextureKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *texture, int textureWidth, int textureHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x 
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int idTextureRgb = id / texturePixels;
		int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
		int idTextureY = idTexturePixel / textureWidth; 
		int idTextureX = (idTexturePixel - idTextureY * textureWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


		if (idTextureRgb < 3) // 3 channels that we will write to
		{
			// the texture is in BGR format, we want RGB
			switch (idTextureRgb)
			{
			case 0: // R
				idTextureRgb = 2; // B
				break;
			case 2: // B
				idTextureRgb = 0; // R
				break;
			}
			// if the texture pixel offset by inputX, inputY, lies inside the target
			if (idTextureX + inputX < targetWidth &&
				idTextureX + inputX >= 0 &&
				idTextureY + inputY < targetHeight &&
				idTextureY + inputY >= 0)
			{
				int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
				int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
				float a = texture[aIndex];
				target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
			}
		}
	}

	/*
	Draws a texture into a 3-component target. RGBA. Checks bounds. Stretches the texture.
	*/
	__global__ void DrawRgbaTextureKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int objectPixels = objectWidth * objectHeight;

		int idObjectRgb = id / objectPixels;
		int idObjectPixel = (id - idObjectRgb * objectPixels); // same as (id % objectPixels), but the kernel runs 10% faster
		int idObjectY = idObjectPixel / objectWidth;
		int idObjectX = (idObjectPixel - idObjectY * objectWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


		if (idObjectRgb < 3) // 3 channels that we will write to
		{
			int targetRgb = idObjectRgb;
			// the texture is in BGR format, we want RGB
			switch (idObjectRgb)
			{
			case 0: // R
				targetRgb = 2; // B
				break;
			case 2: // B
				targetRgb = 0; // R
				break;
			}
			// if the object pixel offset by inputX, inputY, lies inside the target
			if (idObjectX + inputX < targetWidth &&
				idObjectX + inputX >= 0 &&
				idObjectY + inputY < targetHeight &&
				idObjectY + inputY >= 0)
			{
				// nearest neighbor texture X,Y:
				int textureX = textureWidth * idObjectX / objectWidth;
				int textureY = textureHeight * idObjectY / objectHeight;
				int textureId = textureY * textureWidth + textureX;
				
				int rgbIndex = textureId + idObjectRgb * texturePixels;
				float textureValue = texture[rgbIndex];

				int tIndex = targetPixels * targetRgb + targetWidth * (idObjectY + inputY) + (idObjectX + inputX);
				int aIndex = textureId + 3 * texturePixels; // the A component of the texture
				float a = texture[aIndex];
				target[tIndex] = target[tIndex] * (1.0f - a) + a * textureValue;
			}
		}
	}

	/*
	Same as DrawRgbaTextureKernelNearestNeighbor, but texture = mask and texture's pixel values are replaced by a single color
	*/
	__global__ void DrawMaskedColorKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight, 
		float r, float g, float b ) // texture = mask
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int objectPixels = objectWidth * objectHeight;

		int idObjectRgb = id / objectPixels;
		int idObjectPixel = (id - idObjectRgb * objectPixels); // same as (id % objectPixels), but the kernel runs 10% faster
		int idObjectY = idObjectPixel / objectWidth;
		int idObjectX = (idObjectPixel - idObjectY * objectWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


		if (idObjectRgb < 3) // 3 channels that we will write to
		{
			int targetRgb = idObjectRgb;
			// the texture is in BGR format, we want RGB
			switch (idObjectRgb)
			{
			case 0: // R
				targetRgb = 2; // B
				break;
			case 2: // B
				targetRgb = 0; // R
				break;
			}
			// if the object pixel offset by inputX, inputY, lies inside the target
			if (idObjectX + inputX < targetWidth &&
				idObjectX + inputX >= 0 &&
				idObjectY + inputY < targetHeight &&
				idObjectY + inputY >= 0)
			{
				// nearest neighbor texture X,Y:
				int textureX = textureWidth * idObjectX / objectWidth;
				int textureY = textureHeight * idObjectY / objectHeight;
				int textureId = textureY * textureWidth + textureX;

				int tIndex = targetPixels * targetRgb + targetWidth * (idObjectY + inputY) + (idObjectX + inputX);
				int aIndex = textureId + 3 * texturePixels; // the A component of the texture
				float a = texture[aIndex];

				if (a > 0) // mask allows color here
				{
					// apply this: target[tIndex] = target[tIndex] * (1.0f - a) + a * color;
					target[tIndex] = target[tIndex] * (1.0f - a);
					switch (idObjectRgb)
					{
					case 0:
						target[tIndex] += a*r;
						break;
					case 1:
						target[tIndex] += a*g;
						break;
					case 2:
					default:
						target[tIndex] += a*b;
						break;
					}
				}
			}
		}
	}

	/*
	Optimized version of DrawRgbaTextureKernel : avoids division operations (~30% speedup)
	The width of the texture is in blockDim.x
	The height of the texture is distributed between blockDim.y and gridDim.x
	*/
	__global__ void DrawRgbaTextureKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *texture, int textureWidth, int textureHeight)
	{
		int id = blockDim.x * blockDim.y * (blockIdx.y * gridDim.x + blockIdx.x)
			+ blockDim.x * threadIdx.y
			+ threadIdx.x; // 2D grid of 2D blocks; block dimension x = texture width; 
		// grid dimension x + block dimension y = texture height

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int idTextureRgb = blockIdx.y;
		int idTexturePixel = (id - idTextureRgb * texturePixels);
		int idTextureY = blockIdx.x * blockDim.y + threadIdx.y;
		int idTextureX = threadIdx.x;


		if (idTextureRgb < 3) // 3 channels that we will write to
		{
			// the texture is in BGR format, we want RGB
			switch (idTextureRgb)
			{
			case 0: // R
				idTextureRgb = 2; // B
				break;
			case 2: // B
				idTextureRgb = 0; // R
				break;
			}
			// if the texture pixel offset by inputX, inputY, lies inside the target
			if (idTextureX + inputX < targetWidth &&
				idTextureX + inputX >= 0 &&
				idTextureY + inputY < targetHeight &&
				idTextureY + inputY >= 0)
			{
				int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
				int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
				float a = texture[aIndex];
				target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
			}
		}
	}

	/*
	Draws an RGB color into the masked area. The color is drawn in each pixel that has non-0 alpha.
	*/
	__global__ void DrawMaskedColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *textureMask, int textureWidth, int textureHeight, float r, float g, float b) 
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int idTextureRgb = id / texturePixels;
		int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
		int idTextureY = idTexturePixel / textureWidth;
		int idTextureX = (idTexturePixel - idTextureY * textureWidth); // same as (id % textureWidth), but the kernel runs another 10% faster

		if (idTextureRgb < 3) // only RGB channels are interesting
		{
			// if the texture pixel offset by inputX, inputY, lies inside the target
			if (idTextureX + inputX < targetWidth &&
				idTextureX + inputX >= 0 &&
				idTextureY + inputY < targetHeight &&
				idTextureY + inputY >= 0)
			{
				int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
				int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
				float a = textureMask[aIndex];

				if (a > 0) // mask allows color here
				{
					switch (idTextureRgb)
					{
					case 0:
						target[tIndex] = r;
						break;
					case 1:
						target[tIndex] = g;
						break;
					case 2:
					default:
						target[tIndex] = b;
						break;
					}
				}
			}
		}
	}

	/*
	Optimized version of DrawMaskedColorKernel : avoids division operations (~30% speedup)
	The width of the texture is in blockDim.x
	The height of the texture is distributed between blockDim.y and gridDim.x
	*/
	__global__ void DrawMaskedColorKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY,
		float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
	{
		int id = blockDim.x * blockDim.y * (blockIdx.y * gridDim.x + blockIdx.x)
			+ blockDim.x * threadIdx.y
			+ threadIdx.x; // 2D grid of 2D blocks; block dimension x = texture width; 
		// grid dimension x + block dimension y = texture height

		int targetPixels = targetWidth * targetHeight;

		int texturePixels = textureWidth * textureHeight;

		int idTextureRgb = blockIdx.y;
		int idTexturePixel = (id - idTextureRgb * texturePixels);
		int idTextureY = blockIdx.x * blockDim.y + threadIdx.y;
		int idTextureX = threadIdx.x;


		if (idTextureRgb < 3) // only RGB channels are interesting
		{
			// if the texture pixel offset by inputX, inputY, lies inside the target
			if (idTextureX + inputX < targetWidth &&
				idTextureX + inputX >= 0 &&
				idTextureY + inputY < targetHeight &&
				idTextureY + inputY >= 0)
			{
				int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
				int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
				float a = textureMask[aIndex];

				if (a > 0) // mask allows color here
				{
					switch (idTextureRgb)
					{
					case 0:
						target[tIndex] = r;
						break;
					case 1:
						target[tIndex] = g;
						break;
					case 2:
					default:
						target[tIndex] = b;
						break;
					}
				}
			}
		}
	}

	/*
	Convert Raw to RGB
	*/
	__global__ void ExtractRawComponentsToRgbKernel(float *target, int inputWidth, int inputHeight)
	{
		int pixelId = blockDim.x*blockIdx.y*gridDim.x
			+ blockDim.x*blockIdx.x
			+ threadIdx.x;

		int imagePixels = inputWidth * inputHeight;

		if (pixelId >= imagePixels)
			return;

		unsigned int* uTarget = (unsigned int*)target;

		for (int i = 2; i >= 0; i--)
		{
			unsigned int component = uTarget[pixelId];
			component = component >> (8 * (2-i)); // 2-i == RGB -> BGR
			component = component & 0xFF;
			target[imagePixels * i + pixelId] = ((float)component)/255.0f;
			__syncthreads();
		}
	}

}
