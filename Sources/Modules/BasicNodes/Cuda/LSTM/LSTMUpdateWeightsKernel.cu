#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"

extern "C"
{
	typedef enum MyBackPropMethod
	{
		SGD = 0,
		RMSProp = 1,
	} MyBackPropMethod;


	__device__ float Clip(float value, float clip)
	{
		return (clip == 0) * value + (clip != 0) * ((value > clip) * clip + (value < -clip) * -clip + (value >= -clip && value <= clip) * value);

		/* avoids thread divergence, equivalent to:
		if (clip == 0)
			return value;
		else if (value > clip)
			return clip;
		else if (value < -clip)
			return -clip;
		else
			return value;
		*/
	}

	__device__ void SGDWeightUpdate(float trainingRate, float momentum, float clipGradient, float *weights, float *weightDeltas, int weightId, float gradient)
	{
		float weightDelta = trainingRate * Clip(gradient, clipGradient) + momentum * weightDeltas[weightId];
		weightDeltas[weightId] = weightDelta;
		weights[weightId] -= weightDelta;
	}

	__device__ void RMSPropWeightUpdate(float trainingRate, float momentum, float smoothingFactor, float clipGradient, float *weights, float *weightDeltas, float *weightMeanSquares, int weightId, float gradient)
	{
		float rmsGradient = Clip(gradient, clipGradient) + momentum * weightDeltas[weightId];
		weightDeltas[weightId] = rmsGradient;
		float weightMeanSquare = smoothingFactor * weightMeanSquares[weightId] + (1.0f - smoothingFactor) * rmsGradient * rmsGradient;
		if (weightMeanSquare != 0)
			rmsGradient /= sqrtf(weightMeanSquare);
		weightMeanSquares[weightId] = weightMeanSquare;
		weights[weightId] -= trainingRate * rmsGradient;
	}


	__global__ void LSTMUpdateGateWeightsKernelBPTT(
		float *inputGateWeights,
		float *inputGateWeightDeltas,
		float *inputGateWeightMeanSquares,
		float *forgetGateWeights,
		float *forgetGateWeightDeltas,
		float *forgetGateWeightMeanSquares,
		float *outputGateWeights,
		float *outputGateWeightDeltas,
		float *outputGateWeightMeanSquares,

		float* outputGateWeightGradient,
		float* inputGateWeightGradient,
		float* forgetGateWeightGradient,

		MyBackPropMethod backPropMethod,
		float trainingRate,
		float momentum,
		float smoothingFactor,
		float clipGradient,

		int inputCount,
		int previousOutputCount,
		int cellsPerBlock
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

		if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
		{
			if (backPropMethod == RMSProp)
			{
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, outputGateWeights, outputGateWeightDeltas, outputGateWeightMeanSquares, weightId, outputGateWeightGradient[weightId]);
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, inputGateWeights, inputGateWeightDeltas, inputGateWeightMeanSquares, weightId, inputGateWeightGradient[weightId]);
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, forgetGateWeights, forgetGateWeightDeltas, forgetGateWeightMeanSquares, weightId, forgetGateWeightGradient[weightId]);
			}
			else
			{
				SGDWeightUpdate(trainingRate, momentum, clipGradient, outputGateWeights, outputGateWeightDeltas, weightId, outputGateWeightGradient[weightId]);
				SGDWeightUpdate(trainingRate, momentum, clipGradient, inputGateWeights, inputGateWeightDeltas, weightId, inputGateWeightGradient[weightId]);
				SGDWeightUpdate(trainingRate, momentum, clipGradient, forgetGateWeights, forgetGateWeightDeltas, weightId, forgetGateWeightGradient[weightId]);
			}
		}
	}


    __global__ void LSTMUpdateCellWeightsKernelBPTT(
		float *cellInputWeights,
		float *cellInputWeightDeltas,
		float *cellInputWeightMeanSquares,

		MyBackPropMethod backPropMethod,
		float trainingRate,
		float momentum,
		float smoothingFactor,
		float clipGradient,

		float *cellInputWeightGradient,

		int inputCount,
		int previousOutputCount
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerCell = inputCount + previousOutputCount + 1;
		int cellStatesCount = previousOutputCount;

		if (weightId < weightsPerCell * cellStatesCount)
		{
			int cellId = weightId / weightsPerCell;
			if (backPropMethod == RMSProp)
			{
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, cellInputWeights, cellInputWeightDeltas, cellInputWeightMeanSquares, weightId, cellInputWeightGradient[weightId]);
			}
			else
			{
				SGDWeightUpdate(trainingRate, momentum, clipGradient, cellInputWeights, cellInputWeightDeltas, weightId, cellInputWeightGradient[weightId]);
			}
		}
	}



	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*
	/*  ORIGINAL FROM KAREL
	*/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/



	__global__ void LSTMUpdateGateWeightsKernel(
		float *input,
		float *previousOutput,
		float *cellStates,
		float *cellStateErrors,
		float *outputGateDeltas,
		float *inputGateWeights,
		float *inputGateWeightDeltas,
		float *inputGateWeightMeanSquares,
		float *forgetGateWeights,
		float *forgetGateWeightDeltas,
		float *forgetGateWeightMeanSquares,
		float *outputGateWeights,
		float *outputGateWeightDeltas,
		float *outputGateWeightMeanSquares,
		float *inputGateWeightsRTRLPartials,
		float *forgetGateWeightsRTRLPartials,

		MyBackPropMethod backPropMethod,
		float trainingRate,
		float momentum,
		float smoothingFactor,
		float clipGradient,

		int inputCount,
		int previousOutputCount,
		int cellsPerBlock
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

		if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
		{
			int fromId = weightId % weightsPerGate;
			int toId = weightId / weightsPerGate;

			//calculate output gate weight gradient
			int isFromInputUnit = fromId >= 0 && fromId < inputCount;
			int isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
			int isPeephole = (fromId >= inputCount + previousOutputCount) && (fromId < inputCount + previousOutputCount + cellsPerBlock);
			int isFromBiasUnit = fromId == (inputCount + previousOutputCount + cellsPerBlock);

			float inputFromWeight = isFromInputUnit * input[isFromInputUnit * fromId]
				+ isFromPreviousOutputUnit * previousOutput[isFromPreviousOutputUnit * (fromId - inputCount)]
				+ isPeephole * cellStates[isPeephole * (toId * cellsPerBlock + (fromId - inputCount - previousOutputCount))]
				+ isFromBiasUnit * 1;
			float outputGateWeightGradient = outputGateDeltas[toId] * inputFromWeight;

			//calculate input and forget gate weight gradients
			float inputGateWeightGradient = 0;
			float forgetGateWeightGradient = 0;
			//loop through cells
			for (int cellId = toId * cellsPerBlock; cellId < (toId + 1) * cellsPerBlock; cellId++)
			{
				inputGateWeightGradient += cellStateErrors[cellId] * inputGateWeightsRTRLPartials[cellId * weightsPerGate + fromId];
				forgetGateWeightGradient += cellStateErrors[cellId] * forgetGateWeightsRTRLPartials[cellId * weightsPerGate + fromId];
			}

			//update gate weights
			if (backPropMethod == RMSProp)
			{
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, outputGateWeights, outputGateWeightDeltas, outputGateWeightMeanSquares, weightId, outputGateWeightGradient);
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, inputGateWeights, inputGateWeightDeltas, inputGateWeightMeanSquares, weightId, inputGateWeightGradient);
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, forgetGateWeights, forgetGateWeightDeltas, forgetGateWeightMeanSquares, weightId, forgetGateWeightGradient);
			}
			else // SGD
			{
				SGDWeightUpdate(trainingRate, momentum, clipGradient, outputGateWeights, outputGateWeightDeltas, weightId, outputGateWeightGradient);
				SGDWeightUpdate(trainingRate, momentum, clipGradient, inputGateWeights, inputGateWeightDeltas, weightId, inputGateWeightGradient);
				SGDWeightUpdate(trainingRate, momentum, clipGradient, forgetGateWeights, forgetGateWeightDeltas, weightId, forgetGateWeightGradient);
			}
		}
	}

	__global__ void LSTMUpdateCellWeightsKernel(
		float *input,
		float *previousOutput,
		float *cellStateErrors,
		float *cellInputWeights,
		float *cellInputWeightDeltas,
		float *cellInputWeightMeanSquares,
		float *cellWeightsRTRLPartials,

		MyBackPropMethod backPropMethod,
		float trainingRate,
		float momentum,
		float smoothingFactor,
		float clipGradient,

		int inputCount,
		int previousOutputCount,
		int cellsPerBlock
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerCell = inputCount + previousOutputCount + 1;

		if (weightId < weightsPerCell * previousOutputCount)
		{
			int cellId = weightId / weightsPerCell;
			if (backPropMethod == RMSProp)
			{
				RMSPropWeightUpdate(trainingRate, momentum, smoothingFactor, clipGradient, cellInputWeights, cellInputWeightDeltas, cellInputWeightMeanSquares, weightId, cellStateErrors[cellId] * cellWeightsRTRLPartials[weightId]);
			}
			else
			{
				SGDWeightUpdate(trainingRate, momentum, clipGradient, cellInputWeights, cellInputWeightDeltas, weightId, cellStateErrors[cellId] * cellWeightsRTRLPartials[weightId]);
			}
		}
	}

}
