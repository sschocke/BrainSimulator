#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>


extern "C"  
{
	__constant__ int D_INPUT_UNITS;
	__constant__ int D_HIDDEN_UNITS;
	__constant__ int D_OUTPUT_UNITS;


	__global__ void InputWeightsRTRLDerivativesKernel(
		float *input,
		float *hiddenActivationDerivatives,
		float *recurrentWeights,
		float *inputWeightRTRLDerivatives,
		float *previousInputWeightRTRLDerivatives
		)
	{
		int partialId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
		{
			int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
			int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
			int to = weightId / D_INPUT_UNITS;
			int from = weightId % D_INPUT_UNITS;
			
			float sum = 0;
			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousInputWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
			}

			inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * input[from] + sum);
		}
	}

	__global__ void RecurrentWeightsRTRLDerivativesKernel(
		float *previousHiddenActivations,
		float *hiddenActivationDerivatives,
		float *recurrentWeights,
		float *recurrentWeightRTRLDerivatives,
		float *previousRecurrentWeightRTRLDerivatives
		)
	{
		int partialId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS)
		{
			int unitId = partialId / (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
			int weightId = partialId % (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
			int to = weightId / D_HIDDEN_UNITS;
			int from = weightId % D_HIDDEN_UNITS;

			float sum = 0;
			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousRecurrentWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_HIDDEN_UNITS) + weightId];
			}

			recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * previousHiddenActivations[from] + sum);
		}
	}
}
