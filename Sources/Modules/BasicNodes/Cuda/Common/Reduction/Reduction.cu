#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

#include "SharedMemory.cuh"

// INTEGER BASED
#include "i_Sum_i.cuh"
#include "i_MinIdx_2i.cuh"
#include "i_MaxIdx_2i.cuh"
#include "i_MinIdxMaxIdx_4i.cuh"

// SINGLE BASED
#include "f_Sum_f.cuh"
#include "f_MinMax_2f.cuh"
#include "f_MinIdx_fi.cuh"
#include "f_MinIdx_ff.cuh"
#include "f_MaxIdx_fi.cuh"
#include "f_MaxIdx_ff.cuh"
#include "f_MinMax_2f.cuh"
#include "f_MinIdxMaxIdx_fifi.cuh"
#include "f_Average_f.cuh"
#include "c_Average_c.cuh"
#include "c_Sum_c.cuh"

// DOT PRODUCT BASED
#include "i_Dot_i.cuh"
#include "f_Dot_f.cuh"
#include "f_Cosine_f.cuh"
#include "c_ComplexDot_c.cuh"

using namespace std;

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d \n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

int randl()
{
	return (rand() << 16) + rand();
}

template<typename R, unsigned int tCnt, bool finalize>
__forceinline__ __device__ void LogStepShared(R* out, volatile R* partials)
{
	const unsigned int tid = threadIdx.x;

	if (tCnt >= 1024)
	{
		if (tid < 512) {
			partials[tid].op(partials[tid + 512]);
		}
		__syncthreads();
	}
	if (tCnt >= 512)
	{
		if (tid < 256) {
			partials[tid].op(partials[tid + 256]);
		}
		__syncthreads();
	}
	if (tCnt >= 256)
	{
		if (tid < 128) {
			partials[tid].op(partials[tid + 128]);
		}
		__syncthreads();
	}
	if (tCnt >= 128) {
		if (tid < 64) {
			partials[tid].op(partials[tid + 64]);
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		if (tCnt >= 64 && tid < 32) { partials[tid].op(partials[tid + 32]); }
		if (tCnt >= 32 && tid < 16) { partials[tid].op(partials[tid + 16]); }
		if (tCnt >= 16 && tid < 8) { partials[tid].op(partials[tid + 8]); }
		if (tCnt >= 8 && tid < 4) { partials[tid].op(partials[tid + 4]); }
		if (tCnt >= 4 && tid < 2) { partials[tid].op(partials[tid + 2]); }
		if (tCnt >= 2 && tid < 1) { partials[tid].op(partials[tid + 1]); }
	}

	if (tid == 0)
	{
		if (finalize) partials[0].finalize(out);
		else *out = partials[0];
	}
}

__device__ int buffer[8192];
__device__ unsigned int barrier = 0;

template<typename R, typename T, unsigned int tCnt>
__forceinline__ __device__ void DReduction(void* rawOut, volatile const void* rawIn, void* tempBuffer, unsigned int size, unsigned int outOff, unsigned int inOff, unsigned int stride, bool segmented)
{
	__syncthreads();

	if (!tempBuffer) tempBuffer = (void*)buffer;

	unsigned int gridDim_x = gridDim.x;
	unsigned int blockIdx_x = blockIdx.x;

	if (segmented)
	{
		gridDim_x = 1;
		blockIdx_x = 0;

		outOff = blockIdx.x;
		inOff = blockIdx.x * size;
	}

	R* out = reinterpret_cast<R*>(tempBuffer);
	volatile const T* in = reinterpret_cast<volatile const T*>(rawIn) + inOff;

	SharedMemory<R> sPartials;
	const unsigned int tid = threadIdx.x;

	R sum;
	for (unsigned int i = stride * (blockIdx_x * tCnt + tid); i < size; i += stride * tCnt * gridDim_x)
	{
		sum.op(in[i], i + inOff);
	}
	sPartials[tid] = sum;
	__syncthreads();

	if (gridDim_x == 1)
	{
		out = reinterpret_cast<R*>(reinterpret_cast<char*>(rawOut) + R::outSize * outOff);
		LogStepShared<R, tCnt, false>(out, sPartials);
		return;
	}
	LogStepShared<R, tCnt, false>(&out[blockIdx_x], sPartials);

	__shared__ bool lastBlock;
	__threadfence();

	if (tid == 0)
	{
		unsigned int ticket = atomicAdd(&barrier, 1);
		lastBlock = (ticket == gridDim_x - 1);
	}
	__syncthreads();

	if (lastBlock)
	{
		R sum;
		for (unsigned int i = tid; i < gridDim_x; i += tCnt)
		{
			sum.op(out[i]);
		}
		sPartials[threadIdx.x] = sum;
		__syncthreads();

		out = reinterpret_cast<R*>(reinterpret_cast<char*>(rawOut) + R::outSize * outOff);
		LogStepShared<R, tCnt, false>(out, sPartials);
		barrier = 0;
	}
}

template<typename R, typename T, unsigned int tCnt>
__global__ void Reduction(void* rawOut, volatile const void* rawIn, void* tempBuffer, unsigned int size, unsigned int outOff, unsigned int inOff, unsigned int stride, bool segmented)
{
	DReduction<R, T, tCnt>(rawOut, rawIn, tempBuffer, size, outOff, inOff, stride, segmented);
}

template<typename R, typename T, unsigned int tCnt>
__forceinline__ __device__ void DDotProduct(void* rawOut, unsigned int outOff, volatile const void* rawIn1, volatile const void* rawIn2, void* tempBuffer, unsigned int size, bool segmented, bool distributed)
{
	__syncthreads();

	if (tempBuffer == nullptr) tempBuffer = (void*)buffer;

	unsigned int gridDim_x = gridDim.x;
	unsigned int blockIdx_x = blockIdx.x;

	R* out = reinterpret_cast<R*>(tempBuffer);
	volatile const T* in1 = reinterpret_cast<volatile const T*>(rawIn1);
	volatile const T* in2 = reinterpret_cast<volatile const T*>(rawIn2);

	if (segmented)
	{
		gridDim_x = 1;
		blockIdx_x = 0;

		in1 = reinterpret_cast<volatile const T*>(rawIn1) + !distributed * (blockIdx.x * size);
		in2 = reinterpret_cast<volatile const T*>(rawIn2) + blockIdx.x * size;

		outOff = blockIdx.x;
	}

	SharedMemory<R> sPartials;
	const unsigned int tid = threadIdx.x;

	R sum;
	for (unsigned int i = blockIdx_x * tCnt + tid; i < size; i += tCnt * gridDim_x)
	{
		sum.op(in1[i], in2[i], i);
	}
	sPartials[tid] = sum;
	__syncthreads();

	if (gridDim_x == 1)
	{
		out = reinterpret_cast<R*>(reinterpret_cast<char*>(rawOut) + R::outSize * outOff);
		LogStepShared<R, tCnt, true>(out, sPartials);
		return;
	}
	LogStepShared<R, tCnt, false>(&out[blockIdx_x], sPartials);

	__shared__ bool lastBlock;
	__threadfence();

	if (tid == 0)
	{
		unsigned int ticket = atomicAdd(&barrier, 1);
		lastBlock = (ticket == gridDim_x - 1);
	}
	__syncthreads();

	if (lastBlock)
	{
		R sum;
		for (unsigned int i = tid; i < gridDim_x; i += tCnt)
		{
			sum.op(out[i]);
		}
		sPartials[threadIdx.x] = sum;
		__syncthreads();

		out = reinterpret_cast<R*>(reinterpret_cast<char*>(rawOut) + R::outSize * outOff);
		LogStepShared<R, tCnt, true>(out, sPartials);
		barrier = 0;
	}
}

template<typename R, typename T, unsigned int tCnt>
__global__ void DotProduct(void* rawOut, unsigned int outOff, volatile const void* rawIn1, volatile const void* rawIn2, void* tempBuffer, unsigned int size, bool segmented, bool distributed)
{
	DDotProduct<R, T, tCnt>(rawOut, outOff, rawIn1, rawIn2, tempBuffer, size, segmented, distributed);
}

template<typename R, typename T>
void ReductionTemplate()
{
	Reduction<R, T, 32> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	Reduction<R, T, 64> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	Reduction<R, T, 128> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	Reduction<R, T, 256> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	Reduction<R, T, 512> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
}

template<typename R, typename T>
void DotProductTemplate()
{
	DotProduct<R, T, 32> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	DotProduct<R, T, 64> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	DotProduct<R, T, 128> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	DotProduct<R, T, 256> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
	DotProduct<R, T, 512> << <0, 0 >> >(0, 0, 0, 0, 0, 0, 0, 0);
}

extern "C"
void InstantiationDummy()
{
	// INTEGER BASED
	ReductionTemplate < i_Sum_i, int >();
	ReductionTemplate < i_MinIdx_2i, int >();
	ReductionTemplate < i_MaxIdx_2i, int >();
	ReductionTemplate < i_MinIdxMaxIdx_4i, int >();

	// SINGLE BASED
	ReductionTemplate < f_Sum_f, float >();
	ReductionTemplate < f_MinMax_2f, float >();
	ReductionTemplate < f_MinIdx_fi, float >();
	ReductionTemplate < f_MinIdx_ff, float >();
	ReductionTemplate < f_MaxIdx_fi, float >();
	ReductionTemplate < f_MaxIdx_ff, float >();
	ReductionTemplate < f_MinIdxMaxIdx_fifi, float >();
	ReductionTemplate < f_Average_f, float >();
	ReductionTemplate < c_Average_c, Complex >();
	ReductionTemplate < c_Sum_c, Complex >();

	// DOT PRODUCT
	DotProductTemplate <i_Dot_i, int >();
	DotProductTemplate <f_Dot_f, float >();
	DotProductTemplate <f_Cosine_f, float >();
	DotProductTemplate <c_ComplexDot_c, Complex>();
}

typedef void(*reduction_type)(void*, volatile const void*, void*, unsigned int, unsigned int, unsigned int, unsigned int, bool);

template<typename R, typename T, const int bCnt>
void TestReduction(reduction_type kernel, const char* name, int repetitions, int sizeMax, int min, int max, float div, bool segmented)
{
	const int w = 20;
	for (int r = 0; r < repetitions; ++r)
	{
		hipEvent_t startGPU, stopGPU;
		HANDLE_ERROR(hipEventCreate(&startGPU));
		HANDLE_ERROR(hipEventCreate(&stopGPU));
		float timeGPU;
		float timeCPU;

		int inSize = randl() % sizeMax + 1;
		int inOff = randl() % inSize;
		int size = randl() % (inSize - inOff) + 1;
		T* d_in, *h_in = new T[inSize];
		HANDLE_ERROR(hipMalloc(&d_in, sizeof(T) * inSize));
		for (int i = 0; i < inSize; ++i)
		{
			h_in[i] = static_cast<T>(randl() % (max - min) + min) / div;
		}
		HANDLE_ERROR(hipMemcpy(d_in, h_in, sizeof(T) * inSize, hipMemcpyHostToDevice));

		int stride = 1;
		if (randl() % 2 == 0) stride = randl() % (32) + 1;

		int outOff = segmented ? bCnt : randl() % 1000;
		R* d_out;
		R* h_out = reinterpret_cast<R*>(new char[R::outSize*(outOff + 1)]);
		R* c_out = reinterpret_cast<R*>(new char[R::outSize*(outOff + 1)]);
		HANDLE_ERROR(hipMalloc(&d_out, R::outSize * (outOff + 1)));
		HANDLE_ERROR(hipMemcpy(d_out, h_out, R::outSize * (outOff + 1), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipEventRecord(startGPU, 0));

		if (segmented)
			kernel << <bCnt, 1024, sizeof(R) * 1024 >> >(d_out, d_in, nullptr, size / bCnt, 0, 0, stride, segmented);
		else
			kernel << <bCnt, 1024, sizeof(R) * 1024 >> >(d_out, d_in, nullptr, size, outOff, inOff, stride, segmented);

		HANDLE_ERROR(hipEventRecord(stopGPU, 0));
		HANDLE_ERROR(hipEventSynchronize(stopGPU));
		HANDLE_ERROR(hipEventElapsedTime(&timeGPU, startGPU, stopGPU));

		HANDLE_ERROR(hipMemcpy(h_out, d_out, R::outSize * (outOff + 1), hipMemcpyDeviceToHost));

		time_t startCPU, stopCPU;
		int cycles = 100000000 / (inSize - inOff) >= 1 ? 100000000 / (inSize - inOff) : 1;
		startCPU = clock();

		if (segmented)
			for (size_t b = 0; b < bCnt; ++b)
			{
				unsigned int chunkSize = size / bCnt;
				unsigned int chunkOffset = b * chunkSize;
				for (size_t c = 0; c < cycles; ++c)
					R::simulate(c_out, h_in, chunkSize, b, chunkOffset, stride);
			}
		else
			for (size_t c = 0; c < cycles; ++c)
				R::simulate(c_out, h_in, size, outOff, inOff, stride);

		stopCPU = clock();
		timeCPU = difftime(stopCPU, startCPU) / cycles;

		cout << "=== Test: " << name << " ===" << endl;
		cout << left << setw(w) << "Speedup" << setw(w) << "GPU time" << setw(w) << "CPU time" << setw(w)
			<< "input size" << setw(w) << "size" << setw(w) << "output offset" << setw(w) << "input offset" << setw(w) << "stride" << endl;
		cout << left << setw(w) << (timeCPU / timeGPU) << setw(w) << timeGPU << setw(w) << timeCPU << setw(w) << inSize << setw(w) << size
			<< setw(w) << outOff << setw(w) << inOff << setw(w) << stride << endl;

		printf("Check (GPU == CPU): \n");
		bool passed = true;
		if (segmented)
			for (size_t b = 0; b < bCnt; b++)
				passed &= R::check(h_out, c_out, b, h_in);
		else
			passed = R::check(h_out, c_out, outOff, h_in);

		printf("------------\n");
		if (passed) printf("|  PASSED  |\n");
		else printf("|  FAILED  |\n");
		printf("------------\n\n");

		HANDLE_ERROR(hipFree(d_in));
		HANDLE_ERROR(hipFree(d_out));

		delete[] h_in;
		delete[] h_out;
		delete[] c_out;
	}
}

typedef void(*dotproduct_type)(void*, unsigned int, volatile const void*, volatile const void*, void*, unsigned int, bool, bool);

template<typename T>
void Randomize(T& t, int min, int max, float div)
{
	t = static_cast<T>(float(randl() % (max - min) + min) / div);
}

template<>
void Randomize(Complex& t, int min, int max, float div)
{
	t.R = static_cast<float>(float(randl() % (max - min) + min) / div);
	t.I = static_cast<float>(float(randl() % (max - min) + min) / div);
}

template<typename R, typename T, const int bCnt>
void TestDotProduct(dotproduct_type kernel, const char* name, int repetitions, int sizeMax, int min, int max, float div, bool segmented, bool distributed)
{
	const int w = 20;
	for (int r = 0; r < repetitions; ++r)
	{
		hipEvent_t startGPU, stopGPU;
		HANDLE_ERROR(hipEventCreate(&startGPU));
		HANDLE_ERROR(hipEventCreate(&stopGPU));
		float timeGPU;
		float timeCPU;

		int size = randl() % sizeMax + 1;
		T* d_in1, *d_in2, *h_in1 = new T[size], *h_in2 = new T[size];
		HANDLE_ERROR(hipMalloc(&d_in1, sizeof(T) * size));
		HANDLE_ERROR(hipMalloc(&d_in2, sizeof(T) * size));
		for (int i = 0; i < size; ++i)
		{
			Randomize<T>(h_in1[i], min, max, div);
			Randomize<T>(h_in2[i], min, max, div);
		}
		HANDLE_ERROR(hipMemcpy(d_in1, h_in1, sizeof(T) * size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_in2, h_in2, sizeof(T) * size, hipMemcpyHostToDevice));

		int outOff = segmented ? bCnt : randl() % 1000;
		R* d_out;
		R* h_out = reinterpret_cast<R*>(new char[R::outSize*(outOff + 1)]);
		R* c_out = reinterpret_cast<R*>(new char[R::outSize*(outOff + 1)]);
		HANDLE_ERROR(hipMalloc(&d_out, R::outSize * (outOff + 1)));
		HANDLE_ERROR(hipMemcpy(d_out, h_out, R::outSize * (outOff + 1), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipEventRecord(startGPU, 0));

		if (segmented)
			kernel << <bCnt, 1024, sizeof(R) * 1024 >> >(d_out, 0, d_in1, d_in2, nullptr, size / bCnt, segmented, distributed);
		else
			kernel << <bCnt, 1024, sizeof(R) * 1024 >> >(d_out, outOff, d_in1, d_in2, nullptr, size, segmented, distributed);

		HANDLE_ERROR(hipEventRecord(stopGPU, 0));
		HANDLE_ERROR(hipEventSynchronize(stopGPU));
		HANDLE_ERROR(hipEventElapsedTime(&timeGPU, startGPU, stopGPU));

		HANDLE_ERROR(hipMemcpy(h_out, d_out, R::outSize * (outOff + 1), hipMemcpyDeviceToHost));

		time_t startCPU, stopCPU;
		int cycles = 100000000 / size >= 1 ? 100000000 / size : 1;
		startCPU = clock();

		if (segmented)
			for (size_t b = 0; b < bCnt; ++b)
			{
				unsigned int chunkSize = size / bCnt;
				unsigned int chunkOffset = b * chunkSize;
				for (int c = 0; c < cycles; ++c)
					R::simulate(c_out, b, h_in1 + chunkOffset, h_in2 + chunkOffset, chunkSize);
			}
		else
			for (size_t c = 0; c < cycles; ++c)
				R::simulate(c_out, outOff, h_in1, h_in2, size);

		stopCPU = clock();
		timeCPU = difftime(stopCPU, startCPU) / cycles;

		cout << "=== Test: " << name << " ===" << endl;
		cout << left << setw(w) << "Speedup" << setw(w) << "GPU time" << setw(w) << "CPU time" << setw(w) << "size"
			<< setw(w) << "output offset" << endl;
		cout << left << setw(w) << (timeCPU / timeGPU) << setw(w) << timeGPU << setw(w) << timeCPU << setw(w) << size
			<< setw(w) << outOff << endl;

		printf("Check (GPU == CPU): \n");
		bool passed = true;
		if (segmented)
			for (size_t b = 0; b < bCnt; b++)
				passed &= R::check(h_out, c_out, b, h_in1, h_in2);
		else
			passed = R::check(h_out, c_out, outOff, h_in1, h_in2);

		printf("------------\n");
		if (passed) printf("|  PASSED  |\n");
		else printf("|  FAILED  |\n");
		printf("------------\n\n");

		HANDLE_ERROR(hipFree(d_in1));
		HANDLE_ERROR(hipFree(d_in2));
		HANDLE_ERROR(hipFree(d_out));

		delete[] h_in1;
		delete[] h_in2;
		delete[] h_out;
		delete[] c_out;
	}
}

int main(int argc, char* argv[])
{
	srand(time(NULL));

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int repetitions = 10;
	int sizeMax = 10000000;

	// INTEGER BASED
	TestReduction<i_Sum_i, int, 10>(Reduction<i_Sum_i, int, 512>, "Reduction i_Sum_i", repetitions, sizeMax, -10, 10, 1, false);
	TestReduction<i_MinIdx_2i, int, 10>(Reduction<i_MinIdx_2i, int, 512>, "Reduction i_MinIdx_2i", repetitions, sizeMax, -10000, 10000, 1, false);
	TestReduction<i_MaxIdx_2i, int, 10>(Reduction<i_MaxIdx_2i, int, 512>, "Reduction i_MaxIdx_2i", repetitions, sizeMax, -10000, 10000, 1, false);
	TestReduction<i_MinIdxMaxIdx_4i, int, 10>(Reduction<i_MinIdxMaxIdx_4i, int, 512>, "Reduction i_MinIdxMaxIdx_4i", repetitions, sizeMax, 0, 10000, 1, false);

	// INTEGER BASED SEGMENTED
	TestReduction<i_Sum_i, int, 10>(Reduction<i_Sum_i, int, 512>, "Reduction i_Sum_i", repetitions, sizeMax, -10, 10, 1, true);
	TestReduction<i_MinIdx_2i, int, 10>(Reduction<i_MinIdx_2i, int, 512>, "Reduction i_MinIdx_2i", repetitions, sizeMax, -10000, 10000, 1, true);
	TestReduction<i_MaxIdx_2i, int, 10>(Reduction<i_MaxIdx_2i, int, 512>, "Reduction i_MaxIdx_2i", repetitions, sizeMax, -10000, 10000, 1, true);
	TestReduction<i_MinIdxMaxIdx_4i, int, 10>(Reduction<i_MinIdxMaxIdx_4i, int, 512>, "Reduction i_MinIdxMaxIdx_4i", repetitions, sizeMax, 0, 10000, 1, true);

	// SINGLE BASED
	TestReduction<f_Sum_f, float, 10>(Reduction<f_Sum_f, float, 512>, "Reduction f_Sum_f", repetitions, sizeMax, -100, 100, 100, false);
	TestReduction<f_MinMax_2f, float, 10>(Reduction<f_MinMax_2f, float, 512>, "Reduction f_MinMax_2f", repetitions, sizeMax, -100000, 100000, 1000, false);
	TestReduction<f_MinIdx_fi, float, 10>(Reduction<f_MinIdx_fi, float, 512>, "Reduction f_MinIdx_fi", repetitions, sizeMax, -100000, 100000, 1000, false);
	TestReduction<f_MaxIdx_fi, float, 10>(Reduction<f_MaxIdx_fi, float, 512>, "Reduction f_MaxIdx_fi", repetitions, sizeMax, -100000, 100000, 1000, false);
	TestReduction<f_MinIdxMaxIdx_fifi, float, 10>(Reduction<f_MinIdxMaxIdx_fifi, float, 512>, "Reduction f_MinIdxMaxIdx_fifi", repetitions, sizeMax, 0, 100000, 1000, false);

	// SINGLE BASED SEGMENTED
	TestReduction<f_Sum_f, float, 10>(Reduction<f_Sum_f, float, 512>, "Reduction f_Sum_f", repetitions, sizeMax, -100, 100, 100, true);
	TestReduction<f_MinMax_2f, float, 10>(Reduction<f_MinMax_2f, float, 512>, "Reduction f_MinMax_2f", repetitions, sizeMax, -100000, 100000, 1000, true);
	TestReduction<f_MinIdx_fi, float, 10>(Reduction<f_MinIdx_fi, float, 512>, "Reduction f_MinIdx_fi", repetitions, sizeMax, -100000, 100000, 1000, true);
	TestReduction<f_MaxIdx_fi, float, 10>(Reduction<f_MaxIdx_fi, float, 512>, "Reduction f_MaxIdx_fi", repetitions, sizeMax, -100000, 100000, 1000, true);
	TestReduction<f_MinIdxMaxIdx_fifi, float, 10>(Reduction<f_MinIdxMaxIdx_fifi, float, 512>, "Reduction f_MinIdxMaxIdx_fifi", repetitions, sizeMax, 0, 100000, 1000, true);

	// DOT PRODUCT
	TestDotProduct<i_Dot_i, int, 10>(DotProduct<i_Dot_i, int, 512>, "DotProduct i_Dot_i", repetitions, sizeMax, -10, 10, 1, false, false);
	TestDotProduct<f_Dot_f, float, 10>(DotProduct<f_Dot_f, float, 512>, "DotProduct f_Dot_f", repetitions, sizeMax, -100, 100, 100, false, false);
	TestDotProduct<f_Cosine_f, float, 10>(DotProduct<f_Cosine_f, float, 512>, "DotProduct f_Cosine_f", repetitions, sizeMax, -100, 100, 100, false, false);
	TestDotProduct<c_ComplexDot_c, Complex, 10>(DotProduct<c_ComplexDot_c, Complex, 512>, "ComplexDotProduct c_ComplexDot_c", repetitions, sizeMax, -100, 100, 100, false, false);

	// DOT PRODUCT SEGMENTED
	TestDotProduct<i_Dot_i, int, 10>(DotProduct<i_Dot_i, int, 512>, "DotProduct i_Dot_i", repetitions, sizeMax, -10, 10, 1, true, false);
	TestDotProduct<f_Dot_f, float, 10>(DotProduct<f_Dot_f, float, 512>, "DotProduct f_Dot_f", repetitions, sizeMax, -100, 100, 100, true, false);
	TestDotProduct<f_Cosine_f, float, 10>(DotProduct<f_Cosine_f, float, 512>, "DotProduct f_Cosine_f", repetitions, sizeMax, -100, 100, 100, true, false);
	TestDotProduct<c_ComplexDot_c, Complex, 10>(DotProduct<c_ComplexDot_c, Complex, 512>, "ComplexDotProduct c_ComplexDot_c", repetitions, sizeMax, -100, 100, 100, true, false);

	return 0;
}
