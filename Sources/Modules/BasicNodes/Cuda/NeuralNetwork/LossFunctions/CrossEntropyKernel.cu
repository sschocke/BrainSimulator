#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "..\Activation\ActivationFunction.cu"

extern "C"
{
	__global__ void CrossEntropyKernel(
		ActivationFunctionEnum actFunc,
		float *neuronInputPtr,
		float *outputPtr,
		float *targetPtr,
		float *deltaPtr,
		float *costPtr,
		int thisLayerSize,
		int batchSize
		)
	{
		extern __shared__ float loss[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int k = tid;

		loss[tid] = 0;
		__syncthreads();

		while (k < thisLayerSize * batchSize)
		{
			if (!isnan(targetPtr[k]))
			{
				// accumulate loss
				if (targetPtr[k])
					loss[tid] -= logf(outputPtr[k]) / batchSize; // exp(output[k])/sum of exps of outputs should be here by this time (use softmax activation)

				// calculate delta, assuming that target is a vector of 0s with exactly one 1
				deltaPtr[k] += (outputPtr[k] - targetPtr[k]);

				// meaning this would be equivalent
				// deltaPtr[k] += outputPtr[k] - (targetPtr[k] == 1);



			}
			k += blockSize;
		}

		// reduction of loss to cost
		if (blockSize >= 1024) { if (tid < 512) { loss[tid] += loss[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { loss[tid] += loss[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { loss[tid] += loss[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { loss[tid] += loss[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) loss[tid] += loss[tid + 32];
			if (blockSize >= 32) loss[tid] += loss[tid + 16];
			if (blockSize >= 16) loss[tid] += loss[tid + 8];
			if (blockSize >= 8) loss[tid] += loss[tid + 4];
			if (blockSize >= 4) loss[tid] += loss[tid + 2];
			if (blockSize >= 2) loss[tid] += loss[tid + 1];
		}
		if (tid == 0)
			*costPtr = loss[0];
	}
}