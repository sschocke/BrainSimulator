#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>
#include "..\Activation\ActivationFunction.cu"

extern "C"
{

	__device__ int indexFromXY (int x, int y, int width)
	{
		return y * width + x;
	}

	__global__ void PoolingForwardKernel (
		float *inputPtr,
		float *outputPtr,
		int *activatedNeuronsPtr,
		int inputWidth, int inputSize,
		int filterWidth, int filterHeight,
		int horStride, int verStride,
		int outputWidth, int outputSize,
		int thisLayerSize
	)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
				+ blockDim.x * blockIdx.x				//blocks preceeding current block
				+ threadIdx.x;

		if (idx < thisLayerSize)
		{

			int depth = idx / outputSize;
			int depthShift = depth * inputSize;


			int inputTileX = (idx % outputSize) % outputWidth;
			int inputTileY = (idx % outputSize) / outputWidth;
			

			int y = inputTileY * verStride;
			int maxY = y;

			int maxX = inputTileX * horStride;

			
			float maxValue = inputPtr[depthShift + indexFromXY(maxX, y, inputWidth)];

			for (int j = 0; j < filterHeight; j++)
			{
				int x = inputTileX * horStride;
				for (int i = 0; i < filterWidth; i++)
				{
					float value = inputPtr[depthShift + indexFromXY(x, y, inputWidth)];
					if (value > maxValue) {
						value = maxValue;
						maxX = x;
						maxY = y;
					}
					++x;
				}
				++y;
			}

			// probably no need to save weighted input (without activation), since pool layer has no activation
			// if it would have, it would be needed to add this
			outputPtr[idx] = maxValue;
			activatedNeuronsPtr[idx] = depthShift + indexFromXY(maxX, maxY, inputWidth);
		}
	}


	__global__ void PoolingBackwardKernel (
		ActivationFunctionEnum inputActFunc,
		float *thisLayerDelta,
		float *inputLayerDelta,
		float *inputWeightedPtr,
		int *activatedNeuronsPtr,
		int thisLayerSize
	)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
				+ blockDim.x * blockIdx.x				//blocks preceeding current block
				+ threadIdx.x;

		if (idx < thisLayerSize)
		{
			int inputIdx = activatedNeuronsPtr[idx];
			inputLayerDelta[inputIdx] += thisLayerDelta[idx] * EvaluateDerivative(inputActFunc, inputWeightedPtr[inputIdx]);
		}
	}
}