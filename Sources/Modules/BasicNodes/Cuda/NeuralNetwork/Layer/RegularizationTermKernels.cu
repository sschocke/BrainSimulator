#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>
#include "..\Activation\ActivationFunction.cu"

// Gaussian regularization coefficient
__constant__ float RegularizationCoefficient;

extern "C"
{
	__global__ void L1TermKernel(
		float *weightPtr,
		float *L1TermPtr,
		int weights
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < weights) { partialSum[tid] += abs(weightPtr[idx]); idx += blockSize; }

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*L1TermPtr = partialSum[0];
	}

	__global__ void L2TermKernel(
		float *weightPtr,
		float *L2TermPtr,
		int weights
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < weights) { partialSum[tid] += weightPtr[idx] * weightPtr[idx]; idx += blockSize; }

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*L2TermPtr = 0.5f * partialSum[0];
	}

	__global__ void GaussianRegularizationKernel(
		float* means,
		float* sigmas,
		int prevLayerSize,
		float* regularizationPtr
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < prevLayerSize / 2)
		{
			float mu_sq = pow(means[idx], 2);
			float sigma_sq = pow(sigmas[idx], 2);
			partialSum[tid] += mu_sq + sigma_sq - log(sigma_sq);
			idx += blockSize;
		}

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*regularizationPtr = partialSum[0];
	}

	__global__ void GaussianRegularizationDeltaKernel(
		int useSigmaConstant,
		ActivationFunctionEnum prevActFunc,
		float* prevWeighedInputPtr,
		float* prevLayerInputPtr,
		float* prevLayerWeights,
		int prevLayerOutputCount,
		float* meanDeltas,
		float* sigmaDeltas
		)
	{
		// i: previous layer output (which is mu, sigma params)
		int weightId = blockDim.x * blockIdx.y * gridDim.x     //rows preceeding current row in grid
			+ blockDim.x * blockIdx.x                               //blocks preceeding current block
			+ threadIdx.x;

		int prevLayerId = weightId % prevLayerOutputCount;
		int prevPrevLayerId = weightId / prevLayerOutputCount;

		int isMean = prevLayerId < prevLayerOutputCount / 2 || useSigmaConstant;
		int isSigma = prevLayerId >= prevLayerOutputCount / 2 && !useSigmaConstant;

		float regularization = isMean * prevLayerWeights[weightId] * powf(prevLayerInputPtr[prevPrevLayerId], 2)
			+ isSigma * (prevLayerWeights[weightId] * powf(prevLayerInputPtr[prevPrevLayerId], 2) - 1.0f / (prevLayerWeights[weightId]));

		meanDeltas[prevLayerId] += isMean * RegularizationCoefficient * regularization * EvaluateDerivative(prevActFunc, prevWeighedInputPtr[prevLayerId]);
		sigmaDeltas[prevLayerId] += isSigma * RegularizationCoefficient * regularization * EvaluateDerivative(prevActFunc, prevWeighedInputPtr[prevLayerId]);
	}

	__global__ void DropoutMaskKernel(
		float *dropoutMaskPtr,
		float dropout,
		int inputSize
		)
	{
		int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (i < inputSize)
		{
			dropoutMaskPtr[i] = dropout > dropoutMaskPtr[i];
			/*if (dropoutMaskPtr[i] > dropout)
				dropoutMaskPtr[i] = 0.0f;
			else
				dropoutMaskPtr[i] = 1.0f;*/
		}
	}
}